#include "hip/hip_runtime.h"
#include "gui.hpp"

namespace nagi {

GUI::GUI(const std::string& windowName, PathTracer& pathTracer):
    pathTracer{ pathTracer }, scene{ pathTracer.scene }, wSize{ pathTracer.window }, gamma{ scene.config.gamma }, totalSpp{ scene.config.spp } {
    glfwSetErrorCallback(glfw_error_callback);
    if (!glfwInit())
        throw std::runtime_error("Error: Failed to initialize GLFW.");

    // GL 3.0 + GLSL 130
    const char* glsl_version = "#version 130";

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    // Disable resizing window
    glfwWindowHint(GLFW_RESIZABLE, GLFW_FALSE);
    window = glfwCreateWindow(wSize.width, wSize.height, windowName.c_str(), nullptr, nullptr);
    if (window == nullptr) {
        throw std::runtime_error("Error: Failed to create GLFW window.");
    }
    glfwMakeContextCurrent(window);
    glfwSwapInterval(1); // Enable vsync

    // Setup Dear ImGui context
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;
    //io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;     // Enable Keyboard Controls
    //io.ConfigFlags |= ImGuiConfigFlags_NavEnableGamepad;      // Enable Gamepad Controls
    ImGui::StyleColorsDark();

    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init(glsl_version);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
        throw std::runtime_error("Error: Failed to initialize GLAD");
    }

    glGenBuffers(1, &pbo); // make & register PBO
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, 4 * sizeof(GLubyte) * wSize.width * wSize.height, NULL, GL_DYNAMIC_DRAW);

    cudaRun(hipGraphicsGLRegisterBuffer(&regesitered_pbo, pbo, hipGraphicsRegisterFlagsWriteDiscard));
}
void GUI::terminate() {
    if (window) {
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
        cudaRun(hipGraphicsUnregisterResource(regesitered_pbo));
        glDeleteBuffers(1, &pbo);

        ImGui_ImplOpenGL3_Shutdown();
        ImGui_ImplGlfw_Shutdown();
        ImGui::DestroyContext();

        glfwDestroyWindow(window);
        glfwTerminate();

        window = nullptr;
    }

    if (devLuminance) {
        cudaRun(hipFree(devLuminance));
        devLuminance = nullptr;
    }
    if (devDenoisedResult1) {
        cudaRun(hipFree(devDenoisedResult1));
        devDenoisedResult1 = nullptr;
    }
    if (devDenoisedResult2) {
        cudaRun(hipFree(devDenoisedResult2));
        devDenoisedResult2 = nullptr;
    }
}

void GUI::render(float delta) {
	if (window) {
		if (glfwWindowShouldClose(window)) {
			terminate();
			return;
		}

        copyToFrameBuffer();

        glfwPollEvents();

        int display_w, display_h;
        glfwGetFramebufferSize(window, &display_w, &display_h);
        glViewport(0, 0, display_w, display_h);
        glClearColor(clearColor.x * clearColor.w, clearColor.y * clearColor.w, clearColor.z * clearColor.w, clearColor.w);
        glClear(GL_COLOR_BUFFER_BIT);

        // display rendering result via OpenGL
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo); // THE MAGIC LINE #1 
        glDrawPixels(wSize.width, wSize.height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);   // THE MAGIC LINE #2

		// Start the Dear ImGui frame
		ImGui_ImplOpenGL3_NewFrame();
		ImGui_ImplGlfw_NewFrame();
		ImGui::NewFrame();

        {
            ImGui::Begin("Preview Control Panel");

            ImGui::Text("Step          : %d.", step);
            static std::list<float> stack;
            stack.push_back(delta);
            if (stack.size() > 10) stack.pop_front();
            float avg = listSum(stack) / stack.size();

            ImGui::Text("Time remaining: %f sec.", avg * (totalSpp - step));

            ImGui::Text("Present"); ImGui::SameLine();
            static const char* items[] = { "Result", "Albedo", "Normal", "Depth"};
            ImGui::Combo("##Present", &present, items, 4);

            if (present != 2) {
                ImGui::Text("Gamma  "); ImGui::SameLine();
                ImGui::SliderFloat("##Gamma", &gamma, 0.0f, 5.0f);
            }
            if (present == 0) {
                ImGui::Text("Denoise"); ImGui::SameLine();
                ImGui::Checkbox("##Denoise", &denoiser);
                if (denoiser) {
                    ImGui::Text("Normal Weight   "); ImGui::SameLine();
                    ImGui::SliderFloat("##SigmaN", &sigmaN, 0.1f, 256.0f);
                    ImGui::Text("Depth Weight    "); ImGui::SameLine();
                    ImGui::SliderFloat("##SigmaZ", &sigmaZ, 0.1f, 10.0f);
                    ImGui::Text("Luminance Weight"); ImGui::SameLine();
                    ImGui::SliderFloat("##SigmaL", &sigmaL, 0.1f, 10.0f);
                }
            }

            ImGui::End();
        }

		ImGui::EndFrame();
		ImGui::Render();

        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        glfwSwapBuffers(window);

        step++;
    }
}


__global__ void kernCopyResultToFrameBuffer(uchar4* pbo, WindowSize window,float gamma, float* buffer, float blend) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= window.pixels) return;

    int py = idx / window.width;
    int px = idx - py * window.width;

    int idx2 = (window.height - py - 1) * window.width + px;
    //int idx2 = idx;

    glm::vec3 color{ buffer[idx * 3], buffer[idx * 3 + 1], buffer[idx * 3 + 2] };
    color /= (1.f + color);
    color = glm::pow(color, glm::vec3{ 1.f / gamma }) * 255.f * (1.f - blend);

    color.x += pbo[idx2].x * blend;
    color.y += pbo[idx2].y * blend;
    color.z += pbo[idx2].z * blend;

    pbo[idx2].x = glm::clamp((unsigned char)color.x, (unsigned char)0, (unsigned char)255);
    pbo[idx2].y = glm::clamp((unsigned char)color.y, (unsigned char)0, (unsigned char)255);
    pbo[idx2].z = glm::clamp((unsigned char)color.z, (unsigned char)0, (unsigned char)255);
    pbo[idx2].w = 0;
}
__global__ void kernCopyAlbedoToFrameBuffer(uchar4* pbo, WindowSize window,float gamma, float* albedo) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= window.pixels) return;

    int py = idx / window.width;
    int px = idx - py * window.width;

    int idx2 = (window.height - py - 1) * window.width + px;
    //int idx2 = idx;

    glm::vec3 color{ albedo[idx * 3], albedo[idx * 3 + 1], albedo[idx * 3 + 2] };
    color = glm::pow(color, glm::vec3{ 1.f / gamma }) * 255.f;

    pbo[idx2].x = glm::clamp((unsigned char)color.x, (unsigned char)0, (unsigned char)255);
    pbo[idx2].y = glm::clamp((unsigned char)color.y, (unsigned char)0, (unsigned char)255);
    pbo[idx2].z = glm::clamp((unsigned char)color.z, (unsigned char)0, (unsigned char)255);
    pbo[idx2].w = 0;
}
__global__ void kernCopyNormalToFrameBuffer(uchar4* pbo, WindowSize window,float* normal) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= window.pixels) return;

    int py = idx / window.width;
    int px = idx - py * window.width;

    int idx2 = (window.height - py - 1) * window.width + px;
    //int idx2 = idx;

    glm::vec3 color{ normal[idx * 3], normal[idx * 3 + 1], normal[idx * 3 + 2] };
    color = (color + 1.f) / 2.f;
    color *= 255.f;

    pbo[idx2].x = glm::clamp((unsigned char)color.x, (unsigned char)0, (unsigned char)255);
    pbo[idx2].y = glm::clamp((unsigned char)color.y, (unsigned char)0, (unsigned char)255);
    pbo[idx2].z = glm::clamp((unsigned char)color.z, (unsigned char)0, (unsigned char)255);
    pbo[idx2].w = 0;
}
__global__ void kernCopyDepthToFrameBuffer(uchar4* pbo, WindowSize window,float gamma, float* depth) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= window.pixels) return;

    int py = idx / window.width;
    int px = idx - py * window.width;

    int idx2 = (window.height - py - 1) * window.width + px;
    //int idx2 = idx;

    float z = depth[idx] * 255.f;
    z = glm::clamp(glm::pow(z, 1.f / gamma), 0.f, 255.f);

    pbo[idx2].x = (unsigned char)z;
    pbo[idx2].y = (unsigned char)z;
    pbo[idx2].z = (unsigned char)z;
    pbo[idx2].w = 0;
}

void GUI::copyToFrameBuffer() {
    cudaRun(hipGraphicsMapResources(1, &regesitered_pbo, NULL));
    cudaRun(hipGraphicsResourceGetMappedPointer((void**)&devFrameBuffer, NULL, regesitered_pbo))

    static bool isDenoised{ denoiser };
    dim3 blocksPerGrid((wSize.pixels + BLOCK_SIZE - 1) / BLOCK_SIZE);

    if (present == 0) { // result
        if (!denoiser) {
            isDenoised = false;
            kernCopyResultToFrameBuffer <<<blocksPerGrid, BLOCK_SIZE>>>(devFrameBuffer, wSize, gamma, pathTracer.devFrameBuf, 0.f);
        }
        else {
            denoise();
            float blend = isDenoised ? 0.5f : 0.f;
            kernCopyResultToFrameBuffer <<<blocksPerGrid, BLOCK_SIZE>>>(devFrameBuffer, wSize, gamma, devDenoisedResult2, blend);
            isDenoised = true;
        }
    }
    else if (present == 1) { // albedo
        isDenoised = false;
        kernCopyAlbedoToFrameBuffer <<<blocksPerGrid, BLOCK_SIZE>>>(devFrameBuffer, wSize, gamma, pathTracer.devAlbedoBuf);
    }
    else if (present == 2) { // normal
        isDenoised = false;
        kernCopyNormalToFrameBuffer <<<blocksPerGrid, BLOCK_SIZE>>>(devFrameBuffer, wSize, pathTracer.devNormalBuf);
    }
    else if (present == 3) { // depth
        isDenoised = false;
        kernCopyDepthToFrameBuffer <<<blocksPerGrid, BLOCK_SIZE>>>(devFrameBuffer, wSize, gamma, pathTracer.devDepthBuf);
    }

    cudaRun(hipDeviceSynchronize());
    cudaRun(hipGraphicsUnmapResources(1, &regesitered_pbo, NULL))
}

__global__ void kernGetY(int pixels, float* result, float* color) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= pixels) return;

    result[idx] = color[idx * 3] * 0.299f + color[idx * 3 + 1] * 0.587f + color[idx * 3 + 2] * 0.114f;
}
__global__ void kernDiscardColor(int pixels, float* result, float* color, float* albedo) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= pixels) return;

    float a1 = albedo[idx * 3] + FLT_EPSILON;
    float a2 = albedo[idx * 3 + 1] + FLT_EPSILON;
    float a3 = albedo[idx * 3 + 2] + FLT_EPSILON;

    result[idx * 3] = color[idx * 3] / a1;
    result[idx * 3 + 1] = color[idx * 3 + 1] / a2;
    result[idx * 3 + 2] = color[idx * 3 + 2] / a3;
}

__global__ void kernRetrieveColor(int pixels, float* result, float* luminance, float* albedo) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= pixels) return;

    float a1 = albedo[idx * 3] + FLT_EPSILON;
    float a2 = albedo[idx * 3 + 1] + FLT_EPSILON;
    float a3 = albedo[idx * 3 + 2] + FLT_EPSILON;

    result[idx * 3] = luminance[idx * 3] * a1;
    result[idx * 3 + 1] = luminance[idx * 3 + 1] * a2;
    result[idx * 3 + 2] = luminance[idx * 3 + 2] * a3;
}

// reference: https://dl.acm.org/doi/10.1145/3105762.3105770
//            http://diglib.eg.org/handle/10.2312/EGGH.HPG10.067-075

//todo: optimization with shared memeory
__global__ void kernBilateralFilter(
    WindowSize window, int dilation, float* denoised, float* origin, float* luminance, float* luminance2, float* normal, float* depth, float sigmaN, float sigmaZ, float sigmaL) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= window.pixels) return;

    int py = idx / window.width;
    int px = idx - py * window.width;

    float variance{ 0.f };
    float var_w{ 0.f };
#pragma unroll
    for (int i = -1; i < 1; i++) {
#pragma unroll
        for (int j = -1; j < 1; j++) {
            int p = window.width * (py + i) + px + j;
            if (p >= 0 && p < window.pixels) {
                float y2 = luminance2[p];
                float y = luminance[p];
                variance += sqrtf(fabsf(y2 - y * y)) * devGaussianKernel3x3[4 + i * 3 + j];
                var_w += devGaussianKernel3x3[4 + i * 3 + j];
            }
        }
    }

    variance /= (1.f + variance) * var_w;
    variance = fmaxf(VAR_EPSILON, variance);

    //float thisL = luminance[idx];
    glm::vec3 thisL{ origin[idx * 3], origin[idx * 3+1], origin[idx * 3+2] };
    glm::vec3 thisN{ normal[idx * 3], normal[idx * 3 + 1], normal[idx * 3 + 2] };
    float thisZ{ depth[idx] };
    glm::vec3 out{ 0.f };
    glm::vec3 wSum{ 0.f };


#pragma unroll
    for (int i = -FILTER_SIZE_HALF; i <= FILTER_SIZE_HALF; i++) {
#pragma unroll
        for (int j = -FILTER_SIZE_HALF; j <= FILTER_SIZE_HALF; j++) {
            int p = window.width * (py + i * dilation) + px + j * dilation;
            if (p >= 0 && p < window.pixels) {
                glm::vec3 c{ origin[p * 3], origin[p * 3 + 1], origin[p * 3 + 2] };
                if (p == idx) {
                    out += c * devGaussianKernel[FILTER_AREA_HALF];
                    wSum += devGaussianKernel[FILTER_AREA_HALF];
                }
                else {
                    glm::vec3 n{ normal[p * 3], normal[p * 3 + 1], normal[p * 3 + 2] };
                    glm::vec3 l{ origin[p * 3], origin[p * 3 + 1], origin[p * 3 + 2] };
                    float z = depth[p];

                    float wn = powf(fmaxf(0.f, glm::dot(thisN, n)), sigmaN);
                    float wz_tmp = -fabsf(thisZ - z) / sigmaZ;
                    glm::vec3 diff{ thisL - l };
                    float sigmaL2 = sigmaL * variance + FLT_EPSILON;
                    glm::vec3 w{ __expf(-fabsf(diff.x) / sigmaL2 + wz_tmp),
                                 __expf(-fabsf(diff.y) / sigmaL2 + wz_tmp), 
                                 __expf(-fabsf(diff.z) / sigmaL2 + wz_tmp) };
                    //w = w * wn / (fabsf(i * dilation) + fabsf(j * dilation));
                    w = w * wn * devGaussianKernel[FILTER_AREA_HALF + i * FILTER_SIZE + j];
                    out += w * c;
                    wSum += w;
                }
            }
        }
    }
    out /= wSum;
    denoised[idx * 3] = out.x;
    denoised[idx * 3 + 1] = out.y;
    denoised[idx * 3 + 2] = out.z;
}

void GUI::denoise() {
    dim3 blocksPerGrid((wSize.pixels + BLOCK_SIZE - 1) / BLOCK_SIZE);
    if (!devLuminance) {
        cudaRun(hipMalloc((void**)&devLuminance, sizeof(float) * wSize.pixels));
    }
    if (!devDenoisedResult1) {
        cudaRun(hipMalloc((void**)&devDenoisedResult1, sizeof(float) * 3 * wSize.pixels));
    }
    if (!devDenoisedResult2) {
        cudaRun(hipMalloc((void**)&devDenoisedResult2, sizeof(float) * 3 * wSize.pixels));
    }
    
    kernDiscardColor<<<blocksPerGrid, BLOCK_SIZE>>>(wSize.pixels, devDenoisedResult1, pathTracer.devFrameBuf, pathTracer.devAlbedoBuf);
    kernGetY<<<blocksPerGrid, BLOCK_SIZE>>>(wSize.pixels, devLuminance, devDenoisedResult1);
    
	kernBilateralFilter<<<blocksPerGrid, BLOCK_SIZE>>>(
		wSize, 1, devDenoisedResult2, devDenoisedResult1, devLuminance, pathTracer.devLumiance2Buf, pathTracer.devCurrentNormalBuf, pathTracer.devCurrentDepthBuf, sigmaN, sigmaZ, sigmaL);
	std::swap(devDenoisedResult1, devDenoisedResult2);
	kernBilateralFilter<<<blocksPerGrid, BLOCK_SIZE>>>(
		wSize, 2, devDenoisedResult2, devDenoisedResult1, devLuminance, pathTracer.devLumiance2Buf, pathTracer.devCurrentNormalBuf, pathTracer.devCurrentDepthBuf, sigmaN, sigmaZ, sigmaL);
	std::swap(devDenoisedResult1, devDenoisedResult2);
	kernBilateralFilter<<<blocksPerGrid, BLOCK_SIZE>>>(
		wSize, 4, devDenoisedResult2, devDenoisedResult1, devLuminance, pathTracer.devLumiance2Buf, pathTracer.devCurrentNormalBuf, pathTracer.devCurrentDepthBuf, sigmaN, sigmaZ, sigmaL);
	std::swap(devDenoisedResult1, devDenoisedResult2);
	kernBilateralFilter<<<blocksPerGrid, BLOCK_SIZE>>>(
		wSize, 8, devDenoisedResult2, devDenoisedResult1, devLuminance, pathTracer.devLumiance2Buf, pathTracer.devCurrentNormalBuf, pathTracer.devCurrentDepthBuf, sigmaN, sigmaZ, sigmaL);
	std::swap(devDenoisedResult1, devDenoisedResult2);

	kernRetrieveColor<<<blocksPerGrid, BLOCK_SIZE>>>(wSize.pixels, devDenoisedResult2, devDenoisedResult1, pathTracer.devAlbedoBuf);
}


}