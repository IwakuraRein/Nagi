#include "hip/hip_runtime.h"
#include "path_tracer.cuh"

#include <thrust/host_vector.h>  
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/copy.h>

namespace nagi {

void PathTracer::initialize() {
	destroyBuffers();
	allocateBuffers();

	hipMemcpy(devObjBuf, scene.objBuf.data(), scene.objBuf.size() * sizeof(Object), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy devObjBuf failed.");
	hipMemcpy(devMtlBuf, scene.mtlBuf.data(), scene.mtlBuf.size() * sizeof(Material), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy devMtlBuf failed.");
	hipMemcpy(devTrigBuf, scene.trigBuf.data(), scene.trigBuf.size() * sizeof(Triangle), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy devTrigBuf failed.");
	hipMemset(devShadeFlags, 0, PIXEL_COUNT);

	dim3 fullBlocksPerGrid((PIXEL_COUNT + BLOCK_SIZE - 1) / BLOCK_SIZE);
	kernInitializeFrameBuffer<<<fullBlocksPerGrid, BLOCK_SIZE>>>(devFrameBuf);
	checkCUDAError("kernInitializeFrameBuffer failed.");
	spp = 0;
}

void PathTracer::allocateBuffers() {
	hipMalloc((void**)&devObjBuf, scene.objBuf.size() * sizeof(Object));
	checkCUDAError("hipMalloc devObjBuf failed.");
	hipMalloc((void**)&devMtlBuf, scene.mtlBuf.size() * sizeof(Material));
	checkCUDAError("hipMalloc devMtlBuf failed.");
	hipMalloc((void**)&devTrigBuf, scene.trigBuf.size() * sizeof(Triangle));
	checkCUDAError("hipMalloc devTrigBuf failed.");
	hipMalloc((void**)&devFrameBuf, sizeof(float) * PIXEL_COUNT * 3);
	checkCUDAError("hipMalloc devFrameBuf failed.");
	hipMalloc((void**)&devRayPool1, PIXEL_COUNT * sizeof(Path));
	checkCUDAError("hipMalloc devRayPool1 failed.");
	hipMalloc((void**)&devRayPool2, PIXEL_COUNT * sizeof(Path));
	checkCUDAError("hipMalloc devRayPool2 failed.");
	hipMalloc((void**)&devResults1, PIXEL_COUNT * sizeof(IntersectInfo));
	checkCUDAError("hipMalloc devResults1 failed.");
	hipMalloc((void**)&devResults2, PIXEL_COUNT * sizeof(IntersectInfo));
	checkCUDAError("hipMalloc devResults2 failed.");
	hipMalloc((void**)&devShadeFlags, PIXEL_COUNT * sizeof(int));
	checkCUDAError("hipMalloc devInitialRayPool failed.");
}

void PathTracer::destroyBuffers() {
	if (devObjBuf) {
		hipFree(devObjBuf);
		checkCUDAError2("hipFree devObjBuf failed.");
	}
	if (devMtlBuf) {
		hipFree(devMtlBuf);
		checkCUDAError2("hipFree devMtlBuf failed.");
	}
	if (devTrigBuf) {
		hipFree(devTrigBuf);
		checkCUDAError2("hipFree devTrigBuf failed.");
	}
	if (devRayPool1) {
		hipFree(devRayPool1);
		checkCUDAError2("hipFree devRayPool1 failed.");
	}
	if (devRayPool2) {
		hipFree(devRayPool2);
		checkCUDAError2("hipFree devRayPool2 failed.");
	}
	if (devResults1) {
		hipFree(devResults1);
		checkCUDAError2("hipFree devResults1 failed.");
	}
	if (devResults2) {
		hipFree(devResults2);
		checkCUDAError2("hipFree devResults2 failed.");
	}
	if (devShadeFlags) {
		hipFree(devShadeFlags);
		checkCUDAError2("hipFree devShadeFlags failed.");
	}
	if (devFrameBuf) {
		hipFree(devFrameBuf);
		checkCUDAError2("hipFree devFrameBuf failed.");
	}
}

PathTracer::~PathTracer() {
	destroyBuffers();
}

// intersection test -> compact rays -> sort rays according to material -> compute color -> compact rays -> intersection test...
void PathTracer::iterate() {
	for (; spp < scene.config.spp; spp++) {
		dim3 fullBlocksPerGrid((PIXEL_COUNT + BLOCK_SIZE - 1) / BLOCK_SIZE);
		kernInitializeRays<<<fullBlocksPerGrid, BLOCK_SIZE>>>(devRayPool1, scene.config.maxBounce, scene.cam);
		checkCUDAError("kernInitializeRays failed.");
		remainingRays = PIXEL_COUNT;
		while (remainingRays > 0) {
			intersectionTest(devRayPool1, devResults1);
			remainingRays = compactRays(devRayPool1, devRayPool2, devResults1, devResults2);
			std::swap(devRayPool1, devRayPool2);
			std::swap(devResults1, devResults2);
			sort(devRayPool1, devRayPool2, devResults1, devResults2);
			std::swap(devRayPool1, devRayPool2);
			std::swap(devResults1, devResults2);
			shade(devResults1, devShadeFlags);
			remainingRays = compactRays(devRayPool1, devRayPool2, devShadeFlags);
		}
	}
}

void PathTracer::intersectionTest(Path* rayPool, IntersectInfo* results) {
	
}

int PathTracer::compactRays(Path* rayPool, Path* compactedRayPool, IntersectInfo* intersectResults, IntersectInfo* compactedIntersectResults) {
	// delete rays that hit nothing. leave zero in the frame buffer
	// thrust here
	return 0;
}
int PathTracer::compactRays(Path* rayPool, Path* compactedRayPool, int* flags) {
	// delete rays that hit light source. write color into the frame buffer
	// thrust here
	return 0;
}

void PathTracer::sort(Path* rayPool, Path* sortedRayPool, IntersectInfo* intersectResults, IntersectInfo* sortedIntersectResults) {
	// sort rays according to materials
	// thrust here
}

void PathTracer::shade(IntersectInfo* intersectResults, int* flags) {
	// compute color and generate new ray direction
	// if ray should terminate, write -1 into the flag array
}

//__global__ void kernInitializeRays(Path* rayPool, int maxBounce, const glm::vec3 camPos, const glm::mat4 invProjViewMat) {
__global__ void kernInitializeRays(Path* rayPool, int maxBounce, const Camera cam) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= PIXEL_COUNT) return;
	Path path;
	path.pixelIdx = idx;
	path.remainingBounces = maxBounce;
	path.ray.origin = cam.position;

	int py = idx / WINDOW_WIDTH;
	int px = idx - py * WINDOW_HEIGHT;

	//todo: add jitter
	//glm::vec3 ndc{ -1.f + px * PIXEL_WIDTH + HALF_PIXEL_WIDTH, -1.f + py * PIXEL_HEIGHT + HALF_PIXEL_HEIGHT, 0.5f };
	//vecTransform(&ndc, cam.invProjectMat*cam.invViewMat);
	//glm::vec3 dir = ndc - cam.position;
	float halfh = tan(cam.fov / 2);
	float halfw = halfh * cam.aspect;
	path.ray.dir = cam.screenOrigin - cam.upDir * (py * cam.pixelHeight + cam.pixelHeight / 2) + cam.rightDir * (px * cam.pixelWidth + cam.pixelWidth / 2);
	path.ray.dir = glm::normalize(path.ray.dir);
	path.ray.invDir = 1.f / path.ray.dir;
	rayPool[idx] = path;
}

__global__ void kernInitializeFrameBuffer(float* frame) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= PIXEL_COUNT) return;
	frame[idx * 3] = 0.f;
	frame[idx * 3 + 1] = 0.f;
	frame[idx * 3 + 2] = 0.f;
}

__global__ void kernTest(Path* rayPool, Triangle* trigBuf, int num, float* frameBuffer) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= PIXEL_COUNT) return;
	Ray r = rayPool[idx].ray;
	glm::vec3 normal;
	glm::vec3 pickedNormal{ 0.f, 0.f, 0.f };
	float dist;
	float minDist{ FLT_MAX };
	bool found{ false };
	for (int i = 0; i < num; i++) {
		Triangle trig = trigBuf[i];
		if (rayBoxIntersect(r, trig.bbox, &dist)) {
			if (rayTrigIntersect(r, trig, &dist, &normal)) {
				if (dist < minDist) {
					minDist = dist;
					pickedNormal = normal;
					found = true;
				}
			}
		}
	}
	if (found) {
		frameBuffer[idx * 3] = (pickedNormal.x + 1.f) / 2.f;
		frameBuffer[idx * 3 + 1] = (pickedNormal.y + 1.f) / 2.f;
		frameBuffer[idx * 3 + 2] = (pickedNormal.z + 1.f) / 2.f;
	}
}

void PathTracer::test1(float* frameBuffer) {
	dim3 fullBlocksPerGrid((PIXEL_COUNT + BLOCK_SIZE - 1) / BLOCK_SIZE);
	kernInitializeRays << <fullBlocksPerGrid, BLOCK_SIZE >> > (devRayPool1, scene.config.maxBounce, scene.cam);
	checkCUDAError("kernInitializeRays failed.");
	kernTest<<<fullBlocksPerGrid, BLOCK_SIZE>>>(devRayPool1, devTrigBuf, scene.trigBuf.size(), devFrameBuf);
	checkCUDAError("kernTest failed.");
	hipMemcpy(frameBuffer, devFrameBuf, sizeof(float) * PIXEL_COUNT * 3, hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy devFrameBuffer failed.");
}

}