#include "hip/hip_runtime.h"
#include "path_tracer.cuh"

#include <thrust/host_vector.h>  
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

namespace nagi {

void PathTracer::initialize() {
	destroyBuffers();
	allocateBuffers();

	hipMemcpy(devObjBuf, scene.objBuf.data(), scene.objBuf.size() * sizeof(Object), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy devObjBuf failed.");
	hipMemcpy(devMtlBuf, scene.mtlBuf.data(), scene.mtlBuf.size() * sizeof(Material), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy devMtlBuf failed.");
	hipMemcpy(devTrigBuf, scene.trigBuf.data(), scene.trigBuf.size() * sizeof(Triangle), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy devTrigBuf failed.");

	// no need to clear frame buffer. kernWriteFrameBuffer() will do this when spp=1
	//dim3 blocksPerGrid((window.pixels + BLOCK_SIZE - 1) / BLOCK_SIZE);
	//kernInitializeFrameBuffer<<<blocksPerGrid, BLOCK_SIZE>>>(window, devFrameBuf);
	//checkCUDAError("kernInitializeFrameBuffer failed.");
}

void PathTracer::allocateBuffers() {
	hipMalloc((void**)&devObjBuf, scene.objBuf.size() * sizeof(Object));
	checkCUDAError("hipMalloc devObjBuf failed.");
	hipMalloc((void**)&devMtlBuf, scene.mtlBuf.size() * sizeof(Material));
	checkCUDAError("hipMalloc devMtlBuf failed.");
	hipMalloc((void**)&devTrigBuf, scene.trigBuf.size() * sizeof(Triangle));
	checkCUDAError("hipMalloc devTrigBuf failed.");
	hipMalloc((void**)&devFrameBuf, sizeof(float) * window.pixels * 3);
	checkCUDAError("hipMalloc devFrameBuf failed.");
	hipMalloc((void**)&devNormalBuf, sizeof(float) * window.pixels * 3);
	checkCUDAError("hipMalloc devNormalBuf failed.");
	hipMalloc((void**)&devAlbedoBuf, sizeof(float) * window.pixels * 3);
	checkCUDAError("hipMalloc devAlbedoBuf failed.");
	hipMalloc((void**)&devDepthBuf, sizeof(float) * window.pixels);
	checkCUDAError("hipMalloc devDepthBuf failed.");
	hipMalloc((void**)&devRayPool1, window.pixels * sizeof(Path));
	checkCUDAError("hipMalloc devRayPool1 failed.");
	hipMalloc((void**)&devRayPool2, window.pixels * sizeof(Path));
	checkCUDAError("hipMalloc devRayPool2 failed.");
	hipMalloc((void**)&devTerminatedRays, window.pixels * sizeof(Path));
	checkCUDAError("hipMalloc devTerminatedRays failed.");
	hipMalloc((void**)&devResults1, window.pixels * sizeof(IntersectInfo));
	checkCUDAError("hipMalloc devResults1 failed.");
	hipMalloc((void**)&devResults2, window.pixels * sizeof(IntersectInfo));
	checkCUDAError("hipMalloc devResults2 failed.");
}

void PathTracer::destroyBuffers() {
	if (devObjBuf) {
		hipFree(devObjBuf);
		checkCUDAError("hipFree devObjBuf failed.");
		devObjBuf = nullptr;
	}
	if (devMtlBuf) {
		hipFree(devMtlBuf);
		checkCUDAError("hipFree devMtlBuf failed.");
		devMtlBuf = nullptr;
	}
	if (devTrigBuf) {
		hipFree(devTrigBuf);
		checkCUDAError("hipFree devTrigBuf failed.");
		devTrigBuf = nullptr;
	}
	if (devRayPool1) {
		hipFree(devRayPool1);
		checkCUDAError("hipFree devRayPool1 failed.");
		devRayPool1 = nullptr;
	}
	if (devRayPool2) {
		hipFree(devRayPool2);
		checkCUDAError("hipFree devRayPool2 failed.");
		devRayPool2 = nullptr;
	}
	if (devTerminatedRays) {
		hipFree(devTerminatedRays);
		checkCUDAError("hipFree v failed.");
		devTerminatedRays = nullptr;
	}
	if (devResults1) {
		hipFree(devResults1);
		checkCUDAError("hipFree devResults1 failed.");
		devResults1 = nullptr;
	}
	if (devResults2) {
		hipFree(devResults2);
		checkCUDAError("hipFree devResults2 failed.");
		devResults2 = nullptr;
	}
	if (devFrameBuf) {
		hipFree(devFrameBuf);
		checkCUDAError("hipFree devFrameBuf failed.");
		devFrameBuf = nullptr;
	}
	if (devNormalBuf) {
		hipFree(devNormalBuf);
		checkCUDAError("hipFree devNormalBuf failed.");
		devNormalBuf = nullptr;
	}
	if (devAlbedoBuf) {
		hipFree(devAlbedoBuf);
		checkCUDAError("hipFree devAlbedoBuf failed.");
		devAlbedoBuf = nullptr;
	}
	if (devDepthBuf) {
		hipFree(devDepthBuf);
		checkCUDAError("hipFree devDepthBuf failed.");
		devDepthBuf = nullptr;
	}
}

PathTracer::~PathTracer() {
	destroyBuffers();
}

// intersection test -> compact rays -> sort rays according to material -> compute color -> compact rays -> intersection test...
void PathTracer::iterate() {
	std::cout << "Start ray tracing..." << std::endl;

	std::chrono::steady_clock::time_point timer1, timer2;
	timer1 = std::chrono::high_resolution_clock::now();
	for (int spp = 1; spp <= scene.config.spp; spp++) {
		std::cout << "  Begin iteration " << spp << ". " << scene.config.spp - spp << " remaining." << std::endl;
		if (printDetails) timer2 = std::chrono::high_resolution_clock::now();
		dim3 blocksPerGrid((window.pixels + BLOCK_SIZE - 1) / BLOCK_SIZE);
		kernInitializeRays<<<blocksPerGrid, BLOCK_SIZE>>>(window, spp, devRayPool1, scene.config.maxBounce, scene.cam);
		checkCUDAError("kernInitializeRays failed.");
		bool firstIntersection = true;
		int remainingRays = window.pixels;
		while (true) {
			remainingRays = intersectionTest(remainingRays);

			//std::cout << remainingRays << " ";
			if (remainingRays <= 0) break;

			//sortRays(remainingRays);

			if (firstIntersection) {
				generateGbuffer(remainingRays, spp);
				firstIntersection = false;
			}

			remainingRays = shade(remainingRays, spp);
			//std::cout << remainingRays << std::endl;
			if (remainingRays <= 0) break;
		}
		writeFrameBuffer(spp);
		terminatedRayNum = 0;
		if (printDetails) {
			float runningTime = std::chrono::duration<float>(std::chrono::high_resolution_clock::now() - timer2).count();
			std::cout << "  Iteration " << spp << " finished. Time cost: " << runningTime << 
				" seconds. Time Remaining: " << runningTime * (scene.config.spp - spp) << " seconds." << std::endl;
		}
	}
	float runningTime = std::chrono::duration<float>(std::chrono::high_resolution_clock::now() - timer1).count();
	std::cout << "Ray tracing finished. Time cost: " << runningTime << " seconds." << std::endl;
}

__global__ void kernTrigIntersectTest(int rayNum, Path* rayPool, int trigIdxStart, int trigIdxEnd, Triangle* trigBuf, IntersectInfo* out) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= rayNum) return;

	Ray r = rayPool[idx].ray;
	glm::vec3 normal, position;
	glm::vec3 pickedNormal{ 0.f, 0.f, 0.f };
	glm::vec2 pickedUV{ 0.f, 0.f };
	glm::vec2 uv;
	int pickedMtlIdx{ -1 };
	float dist;
	float minDist{ FLT_MAX };
	for (int i = trigIdxStart; i <= trigIdxEnd; i++) {
		Triangle trig = trigBuf[i];
		if (rayBoxIntersect(r, trig.bbox, &dist)) {
			if (rayTrigIntersect(r, trig, &dist, &normal, &uv)) {
				if (dist > 0.f && dist < minDist) {
					minDist = dist;
					pickedNormal = normal;
					pickedUV = uv;
					pickedMtlIdx = trig.mtlIdx;
				}
			}
		}
	}
	IntersectInfo result;
	result.mtlIdx = pickedMtlIdx;
	result.normal = pickedNormal;
	result.uv = pickedUV;
	result.position = r.origin + r.dir * (minDist - 0.001f);
	rayPool[idx].lastHit = pickedMtlIdx; // if pickedMtlIdx >=0, ray hits something
	out[idx] = result;
}

__global__ void kernObjIntersectTest(int rayNum, Path* rayPool, int objNum, Object* objBuf, Triangle* trigBuf, IntersectInfo* out) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= rayNum) return;

	Ray r = rayPool[idx].ray;
	glm::vec3 normal, position;
	glm::vec3 pickedNormal{ 0.f, 0.f, 0.f };
	glm::vec2 pickedUV{ 0.f, 0.f };
	glm::vec2 uv;
	int pickedMtlIdx{ -1 };
	float dist;
	float minDist{ FLT_MAX };
	for (int i = 0; i < objNum; i++) {
		Object obj = objBuf[i];
		if (rayBoxIntersect(r, obj.bbox, &dist)) {
			for (int j = obj.trigIdxStart; j <= obj.trigIdxEnd; j++) {
				Triangle trig = trigBuf[j];
				if (rayBoxIntersect(r, trig.bbox, &dist)) {
					if (rayTrigIntersect(r, trig, &dist, &normal, &uv)) {
						if (dist > 0.f && dist < minDist) {
							minDist = dist;
							pickedNormal = normal;
							pickedUV = uv;
							pickedMtlIdx = trig.mtlIdx;
						}
					}
				}
			}
		}
	}
	IntersectInfo result;
	result.mtlIdx = pickedMtlIdx;
	result.normal = pickedNormal;
	result.uv = pickedUV;
	result.position = r.origin + r.dir * (minDist - 0.001f);
	rayPool[idx].lastHit = pickedMtlIdx; // if pickedMtlIdx >=0, ray hits something
	out[idx] = result;

}

__global__ void kernBVHIntersectTest(int rayNum, Path* rayPool, int rootIdx, BVH::Node* treeBuf, int* treeTrigIdx, Triangle* trigBuf, IntersectInfo* out) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= rayNum) return;

	Ray r = rayPool[idx].ray;
	glm::vec3 normal, position;
	glm::vec3 pickedNormal{ 0.f, 0.f, 0.f };
	glm::vec2 pickedUV{ 0.f, 0.f };
	glm::vec2 uv;
	int pickedMtlIdx{ -1 };
	float dist;
	float minDist{ FLT_MAX };

	BVH::Node stack[MAX_TREE_DEPTH + 1];
	int searchedChildern[MAX_TREE_DEPTH + 1] = { 0 };
	int ptr = 0;
	stack[0] = treeBuf[rootIdx]; // root node;
	while (ptr >= 0) {
		BVH::Node& node = stack[ptr];
		if (node.trigIdxStart >= 0) { // leaf node
			for (int i = node.trigIdxStart; i < node.trigIdxStart + node.size; i++) {
				Triangle trig = trigBuf[treeTrigIdx[i]];
				if (rayBoxIntersect(r, trig.bbox, &dist)) {
					if (rayTrigIntersect(r, trig, &dist, &normal, &uv)) {
						if (dist > 0.f && dist < minDist) {
							minDist = dist;
							pickedNormal = normal;
							pickedUV = uv;
							pickedMtlIdx = trig.mtlIdx;
						}
					}
				}
			}
			searchedChildern[ptr] = 0;
			ptr--;/*
			pickedMtlIdx = node.idx;*/
		}

		else if (node.size > searchedChildern[ptr]) {
			BVH::Node& child = treeBuf[node.children[searchedChildern[ptr]]];
			searchedChildern[ptr]++;
			if (rayBoxIntersect(r, child.bbox, &dist)) {
				stack[ptr + 1] = child;
				ptr++;
				searchedChildern[ptr]=0;
			}
		}
		else {
			searchedChildern[ptr] = 0;
			ptr--;
		}
	}

	IntersectInfo result;
	result.mtlIdx = pickedMtlIdx;
	result.normal = pickedNormal;
	result.uv = pickedUV;
	result.position = r.origin + r.dir * (minDist - 0.001f);
	rayPool[idx].lastHit = pickedMtlIdx; // if pickedMtlIdx >=0, ray hits something
	out[idx] = result;
}

int PathTracer::intersectionTest(int rayNum) {
	dim3 blocksPerGrid((rayNum + BLOCK_SIZE - 1) / BLOCK_SIZE);
	//kernTrigIntersectTest <<<blocksPerGrid, BLOCK_SIZE>>>(rayNum, devRayPool1, 0, scene.trigBuf.size()-1, devTrigBuf, devResults1);
	//kernObjIntersectTest <<<blocksPerGrid, BLOCK_SIZE>>>(rayNum, devRayPool1, scene.objBuf.size(), devObjBuf, devTrigBuf, devResults1);
	kernBVHIntersectTest<<<blocksPerGrid, BLOCK_SIZE>>>(rayNum, devRayPool1, bvh.rootIdx, bvh.devTree, bvh.devTreeTrigIdx, devTrigBuf, devResults1);
	checkCUDAError("kernBVHIntersectTest failed.");

	rayNum = compactRays(rayNum, devRayPool1, devRayPool2, devResults1, devResults2);

	std::swap(devRayPool1, devRayPool2);
	std::swap(devResults1, devResults2);

	return rayNum;
}

// sort rays according to materials
void PathTracer::sortRays(int rayNum) {
	thrust::device_ptr<Path> tRays{ devRayPool1 };
	thrust::device_ptr<IntersectInfo> tResults{ devResults1 };

	thrust::stable_sort_by_key(tResults, tResults+rayNum, tRays, IntersectionComp());
	checkCUDAError("thrust::stable_sort_by_key failed.");
	thrust::stable_sort(tResults, tResults+rayNum, IntersectionComp());
	checkCUDAError("thrust::stable_sort failed.");
}

__global__ void kernGenerateGbuffer(
	int rayNum, float currentSpp, Path* rayPool, IntersectInfo* intersections, Material* mtlBuf, float* albedoBuf, float* normalBuf, float* depthBuf) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= rayNum) return;

	Path p = rayPool[idx];
	int pixel = p.pixelIdx;
	IntersectInfo intersect = intersections[idx];
	intersect.normal /= currentSpp;

	Material mtl = mtlBuf[intersect.mtlIdx];
	glm::vec3 albedo;
	if (hasTexture(mtl, TEXTURE_TYPE_BASE)) {
		float4 baseTex = tex2D<float4>(mtl.baseTex.devTexture, intersect.uv.x, intersect.uv.y);
		albedo = glm::vec3{ baseTex.x, baseTex.y, baseTex.z };
	}
	else albedo = mtl.albedo;
	albedo /= currentSpp;

	float depth = glm::length(intersect.position - p.ray.origin);
	depth /= currentSpp;

	// blend the gbuffer is good for denoising. 
	// reference: https://github.com/tunabrain/tungsten/issues/69
	normalBuf[pixel * 3]     *= (currentSpp - 1.f) / currentSpp;
	normalBuf[pixel * 3 + 1] *= (currentSpp - 1.f) / currentSpp;
	normalBuf[pixel * 3 + 2] *= (currentSpp - 1.f) / currentSpp;
	normalBuf[pixel * 3]     += intersect.normal.x;
	normalBuf[pixel * 3 + 1] += intersect.normal.y;
	normalBuf[pixel * 3 + 2] += intersect.normal.z;
	albedoBuf[pixel * 3]     *= (currentSpp - 1.f) / currentSpp;
	albedoBuf[pixel * 3 + 1] *= (currentSpp - 1.f) / currentSpp;
	albedoBuf[pixel * 3 + 2] *= (currentSpp - 1.f) / currentSpp;
	albedoBuf[pixel * 3]     += albedo.x;
	albedoBuf[pixel * 3 + 1] += albedo.y;
	albedoBuf[pixel * 3 + 2] += albedo.z;
	depthBuf[pixel] *= (currentSpp - 1.f) / currentSpp;
	depthBuf[pixel] += depth;
}
void PathTracer::generateGbuffer(int rayNum, int spp) {
	dim3 blocksPerGrid((rayNum + BLOCK_SIZE - 1) / BLOCK_SIZE);
	kernGenerateGbuffer<<<blocksPerGrid, BLOCK_SIZE>>>(
		rayNum, (float)spp, devRayPool1, devResults1, devMtlBuf, devAlbedoBuf, devNormalBuf, devDepthBuf);
}


// compute color and generate new ray direction
__global__ void kernShading(int rayNum, int spp, Path* rayPool, IntersectInfo* intersections, Material* mtlBuf) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= rayNum) return;

	Path p = rayPool[idx];
	IntersectInfo intersection = intersections[idx];
	Material mtl = mtlBuf[intersection.mtlIdx];

	p.ray.origin = intersection.position;


	if (p.remainingBounces == 0) {
		p.lastHit = -1;
	}
	else {
		if (mtl.type == MTL_TYPE_LIGHT_SOURCE) {
			p.color = p.color * mtl.albedo;
			p.remainingBounces = 0;
		}
		else {
			auto rnd = makeSeededRandomEngine(spp, idx, 0);
			if (mtl.type == MTL_TYPE_OPAQUE) {
				if (glm::dot(intersection.normal, p.ray.dir) >= 0.f) {
					p.lastHit = -1;
					p.remainingBounces = 0;
				}
				else {
					float pdf;
					glm::vec3 wo;
					if (mtl.roughness > 0.9f)
						wo = cosHemisphereSampler(intersection.normal, &pdf, rnd);
					else
						wo = GGXImportanceSampler(mtl.roughness, p.ray.dir, intersection.normal, &pdf, rnd);
					if (glm::dot(wo, intersection.normal) < 0.f) {
						p.lastHit = -1;
						p.remainingBounces = 0;
					}
					else {
						glm::vec3 bsdf = microFacetBrdf(p.ray.dir, wo, intersection.uv, intersection.normal, mtl);
						p.color = p.color * bsdf / (pdf + FLT_EPSILON); // lambert is timed inside the bsdf
						p.ray.dir = wo;
						p.ray.invDir = 1.f / wo;
					}
				}
			}
			//if (mtl.type == MTL_TYPE_TRANSPARENT) {
			// }

			p.remainingBounces--;
		}
	}
	rayPool[idx] = p;
}

int PathTracer::shade(int rayNum, int spp) {
	dim3 blocksPerGrid((rayNum + BLOCK_SIZE - 1) / BLOCK_SIZE);
	kernShading <<<blocksPerGrid, BLOCK_SIZE>>>(rayNum, spp, devRayPool1, devResults1, devMtlBuf);
	checkCUDAError("kernShading failed.");

	rayNum = compactRays(rayNum, devRayPool1, devRayPool2);
	std::swap(devRayPool1, devRayPool2);
	return rayNum;
}

// delete rays that hit nothing
int PathTracer::compactRays(int rayNum, Path* rayPool, Path* compactedRayPool, IntersectInfo* intersectResults, IntersectInfo* compactedIntersectResults) {
	thrust::device_ptr<Path> tRaysIn{ rayPool };
	thrust::device_ptr<Path> tRaysOut{ compactedRayPool };
	thrust::device_ptr<Path> tTerminated{ devTerminatedRays };
	thrust::device_ptr<IntersectInfo> tResultIn{ intersectResults };
	thrust::device_ptr<IntersectInfo> tResultOut{ compactedIntersectResults };

	thrust::device_ptr<Path> tmp = thrust::copy_if(tRaysIn, tRaysIn + rayNum, tRaysOut, ifHit());
	checkCUDAError("thrust::copy_if failed.");
	thrust::copy_if(tRaysIn, tRaysIn + rayNum, tTerminated + terminatedRayNum, ifNotHit());
	checkCUDAError("thrust::copy_if failed.");
	thrust::copy_if(tResultIn, tResultIn + rayNum, tRaysIn, tResultOut, ifHit());
	checkCUDAError("thrust::copy_if failed.");

	int remaining = tmp - tRaysOut;
	terminatedRayNum += (rayNum - remaining);
	return remaining;
	//return 0;
}

// delete terminated rays
int PathTracer::compactRays(int rayNum, Path* rayPool, Path* compactedRayPool) {
	thrust::device_ptr<Path> tRaysIn{ rayPool };
	thrust::device_ptr<Path> tRaysOut{ compactedRayPool };
	thrust::device_ptr<Path> tTerminated{ devTerminatedRays };

	thrust::device_ptr<Path> tmp = thrust::copy_if(tRaysIn, tRaysIn + rayNum, tRaysOut, ifNotTerminated());
	thrust::copy_if(tRaysIn, tRaysIn + rayNum, tTerminated + terminatedRayNum, ifTerminated());
	checkCUDAError("thrust::copy_if failed.");

	int remaining = tmp - tRaysOut;
	terminatedRayNum += (rayNum - remaining);
	return remaining;
	//return 0;
}

__global__ void kernInitializeRays(WindowSize window, int spp, Path* rayPool, int maxBounce, const Camera cam, bool jitter) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= window.pixels) return;
	float rnd1 = 0.0;
	float rnd2 = 0.0;
	if (jitter) {
		thrust::default_random_engine rng = makeSeededRandomEngine(spp, idx, 0);
		thrust::uniform_real_distribution<double> u01(-0.5f, 0.5f);
		rnd1 = u01(rng);
		rnd2 = u01(rng);
	}

	Path path;
	path.pixelIdx = idx;
	path.remainingBounces = maxBounce;
	path.ray.origin = cam.position;

	int py = idx / window.width;
	int px = idx - py * window.width;

	//glm::vec3 ndc{ -1.f + px * PIXEL_WIDTH + HALF_PIXEL_WIDTH, -1.f + py * PIXEL_HEIGHT + HALF_PIXEL_HEIGHT, 0.5f };
	//vecTransform(&ndc, cam.invProjectMat*cam.invViewMat);
	//glm::vec3 dir = ndc - cam.position;
	path.ray.dir = cam.screenOrigin
		- cam.upDir * ((float)py + rnd1) * cam.pixelHeight + cam.halfPixelHeight
		+ cam.rightDir * ((float)px + rnd2) * cam.pixelWidth + cam.halfPixelWidth;
	path.ray.dir = glm::normalize(path.ray.dir);
	path.ray.invDir = 1.f / path.ray.dir;
	path.lastHit = 1;
	path.color = glm::vec3{ 1.f, 1.f, 1.f };
	rayPool[idx] = path;
}

__global__ void kernInitializeFrameBuffer(WindowSize window, float* frame) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= window.pixels) return;
	frame[idx * 3] = 0.f;
	frame[idx * 3 + 1] = 0.f;
	frame[idx * 3 + 2] = 0.f;
}

__global__ void kernWriteFrameBuffer(WindowSize window, float currentSpp, Path* rayPool, float* frameBuffer) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= window.pixels) return;

	Path path = rayPool[idx];
	if (path.lastHit < 0) { // ray didn't hit anything, or didn't hit light source in the end
		path.color = glm::vec3{ 0.f };
	}
	frameBuffer[path.pixelIdx * 3]     *= (currentSpp - 1.f) / currentSpp;
	frameBuffer[path.pixelIdx * 3 + 1] *= (currentSpp - 1.f) / currentSpp;
	frameBuffer[path.pixelIdx * 3 + 2] *= (currentSpp - 1.f) / currentSpp;
	frameBuffer[path.pixelIdx * 3]     += (path.color.x / currentSpp);
	frameBuffer[path.pixelIdx * 3 + 1] += (path.color.y / currentSpp);
	frameBuffer[path.pixelIdx * 3 + 2] += (path.color.z / currentSpp);
}

void PathTracer::writeFrameBuffer(int spp) {
	dim3 blocksPerGrid((window.pixels + BLOCK_SIZE - 1) / BLOCK_SIZE);
	kernWriteFrameBuffer <<<blocksPerGrid, BLOCK_SIZE>>>(window, (float)scene.config.spp, devTerminatedRays, devFrameBuf);

}

//__global__ void kernTest(Path* rayPool, Triangle* trigBuf, int num, float* frameBuffer) {
//	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
//	if (idx >= window.pixels) return;
//	Ray r = rayPool[idx].ray;
//	glm::vec3 normal;
//	glm::vec3 pickedNormal{ 0.f, 0.f, 0.f };
//	float dist;
//	float minDist{ FLT_MAX };
//	bool found{ false };
//	for (int i = 0; i < num; i++) {
//		Triangle trig = trigBuf[i];
//		if (rayBoxIntersect(r, trig.bbox, &dist)) {
//			if (rayTrigIntersect(r, trig, &dist, &normal)) {
//				if (dist < minDist) {
//					minDist = dist;
//					pickedNormal = normal;
//					found = true;
//				}
//			}
//		}
//	}
//	if (found) {
//		frameBuffer[idx * 3] = (pickedNormal.x + 1.f) / 2.f;
//		frameBuffer[idx * 3 + 1] = (pickedNormal.y + 1.f) / 2.f;
//		frameBuffer[idx * 3 + 2] = (pickedNormal.z + 1.f) / 2.f;
//	}
//}

std::unique_ptr<float[]> PathTracer::getFrameBuffer() {
	if (devFrameBuf) {
		std::unique_ptr<float[]> ptr{ new float[window.pixels * 3] };
		hipMemcpy(ptr.get(), devFrameBuf, window.pixels * 3 * sizeof(float), hipMemcpyDeviceToHost);
		return ptr;
	}
	else {
		throw std::runtime_error("Error: Frame buffer isn't allocated yet.");
	}
}
void PathTracer::copyFrameBuffer(float* frameBuffer) {
	if (devFrameBuf) {
		hipMemcpy(frameBuffer, devFrameBuf, window.pixels * 3 * sizeof(float), hipMemcpyDeviceToHost);
	}
	else {
		throw std::runtime_error("Error: Frame buffer isn't allocated yet.");
	}
}
std::unique_ptr<float[]> PathTracer::getNormalBuffer() {
	if (devNormalBuf) {
		std::unique_ptr<float[]> ptr{ new float[window.pixels * 3] };
		hipMemcpy(ptr.get(), devNormalBuf, window.pixels * 3 * sizeof(float), hipMemcpyDeviceToHost);
		return ptr;
	}
	else {
		throw std::runtime_error("Error: Normal buffer isn't allocated yet.");
	}
}
std::unique_ptr<float[]> PathTracer::getAlbedoBuffer() {
	if (devAlbedoBuf) {
		std::unique_ptr<float[]> ptr{ new float[window.pixels * 3] };
		hipMemcpy(ptr.get(), devAlbedoBuf, window.pixels * 3 * sizeof(float), hipMemcpyDeviceToHost);
		return ptr;
	}
	else {
		throw std::runtime_error("Error: Albedo buffer isn't allocated yet.");
	}
}
std::unique_ptr<float[]> PathTracer::getDepthBuffer() {
	if (devDepthBuf) {
		std::unique_ptr<float[]> ptr{ new float[window.pixels] };
		hipMemcpy(ptr.get(), devDepthBuf, window.pixels * sizeof(float), hipMemcpyDeviceToHost);
		return ptr;
	}
	else {
		throw std::runtime_error("Error: Depth buffer isn't allocated yet.");
	}
}

}