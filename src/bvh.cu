#include "hip/hip_runtime.h"
#include "bvh.cuh"

namespace nagi {

BVH::~BVH() {
	if (devTree) {
		hipFree(devTree);
		checkCUDAError("hipFree devTree failed.");
	}
	if (devTreeTrigIdx) {
		hipFree(devTreeTrigIdx);
		checkCUDAError("hipFree devTree failed.");
	}
}

void BVH::build() {
	std::chrono::steady_clock::time_point timer;
	timer = std::chrono::high_resolution_clock::now();
	std::cout << "Building BVH... ";

	trigIndices = std::make_shared<std::list<int>>();
	auto initialIndices = std::make_shared<std::list<int>>();
	initialIndices->resize(scene.trigBuf.size());
	int i = 0;
	for (auto it = initialIndices->begin(); it != initialIndices->end(); it++) {
		*it = i;
		i++;
	}

	rootIdx = buildNode(0, initialIndices, scene.bbox);

	hipMalloc((void**)&devTreeTrigIdx, sizeof(int) * trigIndices->size());
	checkCUDAError("hipMalloc devTreeTrigIdx failed.");

	i = 0;
	for (auto it = trigIndices->begin(); it != trigIndices->end(); it++) {
		hipMemcpy(devTreeTrigIdx + i, &(*it), sizeof(int), hipMemcpyHostToDevice);
		i++;
	}
	checkCUDAError("hipMemcpy devTreeTrigIdx failed.");

	hipMalloc((void**)&devTree, sizeof(Node) * tree.size());
	checkCUDAError("hipMalloc devTree failed.");

	hipMemcpy(devTree, tree.data(), sizeof(Node) * tree.size(), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy devTree failed.");

	float runningTime = std::chrono::duration<float>(std::chrono::high_resolution_clock::now() - timer).count();
	std::cout << "Done. Tirangles in BVH: " << tree.size() << ". Time cost : " << runningTime << " seconds." << std::endl;
}

int BVH::buildNode(
	int layer, std::shared_ptr<std::list<int>> trigs, BoundingBox bbox) {
	if (trigs->size() == 0) return -1;
	if (trigs->size() > TERMINATE_NUM && layer != MAX_TREE_DEPTH) {

		glm::vec3 eps{ FLT_EPSILON, FLT_EPSILON, FLT_EPSILON };
		eps = glm::max(eps, bbox.halfExtent * 0.01f);
		glm::vec3 halfX = glm::vec3{ bbox.halfExtent.x, 0.f, 0.f };
		glm::vec3 halfY = glm::vec3{ 0.f, bbox.halfExtent.y, 0.f };
		glm::vec3 halfZ = glm::vec3{ 0.f, 0.f, bbox.halfExtent.z };

		// prepare children's bounding boxes

		BoundingBox b0{}; updateBoundingBox(bbox.min, bbox.center, &b0);
		BoundingBox b1{}; updateBoundingBox(b0.min + halfZ, b0.max + halfZ, &b1);
		BoundingBox b2{}; updateBoundingBox(b0.min + halfY, b0.max + halfY, &b2);
		BoundingBox b3{}; updateBoundingBox(b0.min + halfY + halfZ, b0.max + halfY + halfZ, &b3);

		BoundingBox b4{}; updateBoundingBox(b0.min + halfX, b0.max + halfX, &b4);
		BoundingBox b5{}; updateBoundingBox(b4.min + halfZ, b4.max + halfZ, &b5);
		BoundingBox b6{}; updateBoundingBox(b4.min + halfY, b4.max + halfY, &b6);
		BoundingBox b7{}; updateBoundingBox(b4.min + halfY + halfZ, b4.max + halfY + halfZ, &b7);
		b0.min -= eps; b0.max += eps;
		b1.min -= eps; b1.max += eps;
		b2.min -= eps; b2.max += eps;
		b3.min -= eps; b3.max += eps;
		b4.min -= eps; b4.max += eps;
		b5.min -= eps; b5.max += eps;
		b6.min -= eps; b6.max += eps;
		b7.min -= eps; b7.max += eps;

		// store children's triangles
		std::shared_ptr<std::list<int>> trigs0{ new std::list<int> };
		std::shared_ptr<std::list<int>> trigs1{ new std::list<int> };
		std::shared_ptr<std::list<int>> trigs2{ new std::list<int> };
		std::shared_ptr<std::list<int>> trigs3{ new std::list<int> };
		std::shared_ptr<std::list<int>> trigs4{ new std::list<int> };
		std::shared_ptr<std::list<int>> trigs5{ new std::list<int> };
		std::shared_ptr<std::list<int>> trigs6{ new std::list<int> };
		std::shared_ptr<std::list<int>> trigs7{ new std::list<int> };

		// find triangles

		for (auto it = trigs->begin(); it != trigs->end(); it++) {
			Triangle t = scene.trigBuf[*it];
			if (tirgBoxIntersect(t, b0)) {
				trigs0->push_back(*it);
			}
			if (tirgBoxIntersect(t, b1)) {
				trigs1->push_back(*it);
			}
			if (tirgBoxIntersect(t, b2)) {
				trigs2->push_back(*it);
			}
			if (tirgBoxIntersect(t, b3)) {
				trigs3->push_back(*it);
			}
			if (tirgBoxIntersect(t, b4)) {
				trigs4->push_back(*it);
			}
			if (tirgBoxIntersect(t, b5)) {
				trigs5->push_back(*it);
			}
			if (tirgBoxIntersect(t, b6)) {
				trigs6->push_back(*it);
			}
			if (tirgBoxIntersect(t, b7)) {
				trigs7->push_back(*it);
			}
		}

		trigs.reset();

		Node node{
			0,
			{0},
			-1,
			bbox
		};

		int child;
		child = buildNode(layer + 1, trigs0, b0);
		if (child >= 0) node.children[node.size++] = child;

		child = buildNode(layer + 1, trigs1, b1);
		if (child >= 0) node.children[node.size++] = child;

		child = buildNode(layer + 1, trigs2, b2);
		if (child >= 0) node.children[node.size++] = child;

		child = buildNode(layer + 1, trigs3, b3);
		if (child >= 0) node.children[node.size++] = child;

		child = buildNode(layer + 1, trigs4, b4);
		if (child >= 0) node.children[node.size++] = child;

		child = buildNode(layer + 1, trigs5, b5);
		if (child >= 0) node.children[node.size++] = child;

		child = buildNode(layer + 1, trigs6, b6);
		if (child >= 0) node.children[node.size++] = child;

		child = buildNode(layer + 1, trigs7, b7);
		if (child >= 0) node.children[node.size++] = child;

		if (node.size > 0) {
			tree.push_back(std::move(node));
			return tree.size() - 1;
		}
		if (node.size == 1) {
			return node.children[0];
		}
		else return -1;
	}
	else {
		// construct a leaf node
		Node node{
			trigs->size(),
			{0},
			trigIndices->size(),
			bbox
		};
		trigIndices->splice(trigIndices->end(), *trigs);
		tree.push_back(std::move(node));
		return tree.size() - 1;
	}
}

}